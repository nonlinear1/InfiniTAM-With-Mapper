#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSceneReconstructionEngine_CUDA.h"
#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMSceneReconstructionEngine.h"
#include "../../../Objects/ITMRenderState_VH.h"
#include "../../../ITMLib.h"

struct AllocationTempData {
	int noAllocatedVoxelEntries;
	int noAllocatedExcessEntries;
	int noVisibleBlocks;
};

using namespace ITMLib::Engine;

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *localVBA,
										  const ITMHashEntry *hashTable,
										  Vector3i *visibleBlockPositions,
										  const Vector4u *rgb,
										  Vector2i rgbImgSize,
										  const float *depth,
										  Vector2i imgSize,
										  Matrix4f M_d,
										  Matrix4f M_rgb,
										  Vector4f projParams_d,
										  Vector4f projParams_rgb,
										  float _voxelSize,
										  float mu,
										  int maxW,
										  ITMLib::Engine::WeightParams weightParams);

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustrum_max, int *locks);

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList,
											   int *excessAllocationList,
											   ITMHashEntry *hashTable,
											   int noTotalEntries,
											   AllocationTempData *allocData,
											   uchar *entriesAllocType,
											   uchar *entriesVisibleType,
											   Vector4s *blockCoords,
											   int currentFrame);

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesVisibleType);

__global__ void setToType3(uchar *entriesVisibleType,
						   Vector3i *visibleBlockPositions,
						   int noVisibleBlocks,
						   ITMHashEntry *hashTable);

template<bool useSwapping>
__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashSwapState *swapStates, int noTotalBlocks,
	Vector3i *visibleBlockPositions, AllocationTempData *allocData, uchar *entriesVisibleType,
	Matrix4f M_d, Vector4f projParams_d, Vector2i depthImgSize, float voxelSize);

/// \brief Erases blocks whose weight is smaller than 'maxWeight', and marks blocks which become
///        empty in the process as pending deallocation in `outBlocksToDeallocate`.
/// \tparam TVoxel The type of voxel representation to operate on (grayscale/color, float/short, etc.)
/// \param localVBA The raw storage where the hash map entries reside.
/// \param hashTable Maps entry IDs to addresses in the local VBA.
/// \param visibleBlockPositions A list of blocks on which to operate (typically, this is the list
///                              containing the visible blocks $k$ frames ago). The size of the list
///                              should be known in advance, and be implicitly range-checked by
///                              setting the grid size's x-dimension to it.
/// \param minAge The minimum age for a voxel to be considered for decay.
/// \param maxWeight All voxels above `minAge` with depth weight smaller than or equal to this are
///                  decayed.
/// \param voxelAllocationList Freelist containing IDs of free voxel blocks in `localVBA`.
/// \param lastFreeBlockId Index of the last free block in `voxelAllocationList`.
/// \param locks Used for locking buckets when performing block deletion.
/// \param currentFrame The index of the current frame being processed by the SLAM system.
/// \param entriesVisibleType Maps hash table indices to an enum symbolizing whether a particular
///                           block is visible.
template<class TVoxel>
__global__ void decay_device(TVoxel *localVBA,
							 ITMHashEntry *hashTable,
							 Vector3i *visibleBlockPositions,
							 int minAge,
							 int maxWeight,
							 int *voxelAllocationList,
							 int *lastFreeBlockId,
							 int *locks,
							 int currentFrame,
							 uchar *entriesVisibleType);

/// \brief Used to perform voxel decay on all voxels in a volume.
/// Similar to `decay_device`, but operates on an entire volume of voxels, instead of a list of
/// blocks visible at some point in time.
template<class TVoxel>
__global__ void decayFull_device(
		const Vector4s *usedBlockPositions,
		TVoxel *localVBA,
		ITMHashEntry *hashTable,
		int minAge,
		int maxWeight,
		int *voxelAllocationList,
		int *lastFreeBlockId,
		int *locks,
		int currentFrame,
		uchar *entriesVisibleType);


// host methods

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMSceneReconstructionEngine_CUDA(
		long sdfLocalBlockNum
) {
	ITMSafeCall(hipMalloc((void**)&allocationTempData_device, sizeof(AllocationTempData)));
	ITMSafeCall(hipHostMalloc((void**)&allocationTempData_host, sizeof(AllocationTempData)));

	int noTotalEntries = ITMVoxelBlockHash::noTotalEntries;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector4s)));

	ITMSafeCall(hipMalloc((void**)&lastFreeBlockId_device, 1 * sizeof(int)));
	ITMSafeCall(hipMalloc(&locks_device, SDF_BUCKET_NUM * sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&allocatedBlockPositions_device, sdfLocalBlockNum * sizeof(Vector4s)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipHostFree(allocationTempData_host));
	ITMSafeCall(hipFree(allocationTempData_device));
	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));

	ITMSafeCall(hipFree(lastFreeBlockId_device));
	ITMSafeCall(hipFree(locks_device));
	ITMSafeCall(hipFree(allocatedBlockPositions_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ResetScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	totalDecayedBlockCount = 0;
	// Clean up the visible frame queue used in voxel decay.
	while (! frameVisibleBlocks.empty()) {
		delete frameVisibleBlocks.front().blockCoords;
		frameVisibleBlocks.pop();
	}

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;

	ITMHashEntry tmpEntry;
	memset(&tmpEntry, 0, sizeof(ITMHashEntry));
	tmpEntry.ptr = -2;
	ITMHashEntry *hashEntry_ptr = scene->index.GetEntries();
	memsetKernel<ITMHashEntry>(hashEntry_ptr, tmpEntry, scene->index.noTotalEntries);
	int *excessList_ptr = scene->index.GetExcessAllocationList();
	fillArrayKernel<int>(excessList_ptr, SDF_EXCESS_LIST_SIZE);

	scene->index.SetLastFreeExcessListId(SDF_EXCESS_LIST_SIZE - 1);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::AllocateSceneFromDepth(
		ITMScene<TVoxel, ITMVoxelBlockHash> *scene,
		const ITMView *view,
		const ITMTrackingState *trackingState,
		const ITMRenderState *renderState,
		bool onlyUpdateVisibleList
) {
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;
	M_d = trackingState->pose_d->GetM(); M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	ITMHashEntry *hashTable = scene->index.GetEntries();
	ITMHashSwapState *swapStates = scene->useSwapping ? scene->globalCache->GetSwapStates(true) : 0;

	// The sum of the nr. of buckets, plus and the excess list size.
	int noTotalEntries = scene->index.noTotalEntries;
	Vector3i *visibleBlockPositions = renderState_vh->GetVisibleBlockPositions();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeVS(256, 1);
	dim3 gridSizeVS((int)ceil((float)renderState_vh->noVisibleBlocks / (float)cudaBlockSizeVS.x));

	float oneOverVoxelSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);

	AllocationTempData *tempData = static_cast<AllocationTempData*>(allocationTempData_host);
	tempData->noAllocatedVoxelEntries = scene->localVBA.lastFreeBlockId;
	tempData->noAllocatedExcessEntries = scene->index.GetLastFreeExcessListId();
	tempData->noVisibleBlocks = 0;
	ITMSafeCall(hipMemcpyAsync(allocationTempData_device, tempData, sizeof(AllocationTempData), hipMemcpyHostToDevice));

	ITMSafeCall(hipMemsetAsync(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));

	if (gridSizeVS.x > 0) {
		// Flags all previously visible blocks accordingly (runs for every element in the
		// visibleEntryIDs list).
		// 0 = invisible
		// 1 = visible and in memory
		// 2 = visible and swapped out
		// 3 = visible at previous frame and in memory
		//
		// Note: we could have a kernel map from visible keys to visible IDs here, or simply pass
		//       the keys to 'setToType3'.
		//
		setToType3<<<gridSizeVS, cudaBlockSizeVS>>>(
				entriesVisibleType,
				visibleBlockPositions,
				renderState_vh->noVisibleBlocks,
				hashTable);
	}

	ITMSafeCall(hipMemset(locks_device, 0, sizeof(int) * SDF_BUCKET_NUM));
	buildHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType,
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max, locks_device);

	bool useSwapping = scene->useSwapping;
	if (onlyUpdateVisibleList) {
		useSwapping = false;
	}

	if (!onlyUpdateVisibleList)
	{
		allocateVoxelBlocksList_device<<<gridSizeAL, cudaBlockSizeAL>>>(
				voxelAllocationList,
				excessAllocationList, hashTable,
				noTotalEntries,
				(AllocationTempData *) allocationTempData_device,
				entriesAllocType_device,
				entriesVisibleType,
				blockCoords_device,
				frameIdx);
	}

	if (useSwapping) {
		buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(
				hashTable,
				swapStates,
				noTotalEntries,
				visibleBlockPositions,
				(AllocationTempData *) allocationTempData_device,
				entriesVisibleType,
				M_d,
				projParams_d,
				depthImgSize,
				voxelSize);
	}
	else {
		buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleBlockPositions,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize);
	}

	if (useSwapping)
	{
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(tempData, allocationTempData_device, sizeof(AllocationTempData), hipMemcpyDeviceToHost));
	renderState_vh->noVisibleBlocks = tempData->noVisibleBlocks;
	scene->localVBA.lastFreeBlockId = tempData->noAllocatedVoxelEntries;
	scene->index.SetLastFreeExcessListId(tempData->noAllocatedExcessEntries);

	int totalBlockCount = scene->index.getNumAllocatedVoxelBlocks();
	size_t visibleBlockCount = static_cast<size_t>(tempData->noVisibleBlocks);
	size_t visibleBlocksByteCount = visibleBlockCount * sizeof(Vector3i);

	// Keep track of the visible blocks, which will be used later by the voxel decay mechanism.
	ORUtils::MemoryBlock<Vector3i> *visibleEntryIDsCopy = nullptr;
	if (visibleBlocksByteCount > 0) {
		visibleEntryIDsCopy = new ORUtils::MemoryBlock<Vector3i>(visibleBlocksByteCount, MEMORYDEVICE_CUDA);
		ITMSafeCall(hipMemcpy(visibleEntryIDsCopy->GetData(MEMORYDEVICE_CUDA),
							   visibleBlockPositions,
							   visibleBlocksByteCount,
							   hipMemcpyDeviceToDevice));
	}
	VisibleBlockInfo visibleBlockInfo = {
		visibleBlockCount,
		frameIdx,
		visibleEntryIDsCopy,
	};
	frameVisibleBlocks.push(visibleBlockInfo);
	frameIdx++;

	// This just returns the size of the pre-allocated buffer.
	long allocatedBlocks = scene->index.getNumAllocatedVoxelBlocks();
	// This is the number of blocks we are using out of the chunk that was allocated initially on
	// the GPU (for non-swapping case).
	long usedBlocks = allocatedBlocks - scene->localVBA.lastFreeBlockId - 1;

	long allocatedExcessEntries = SDF_EXCESS_LIST_SIZE;
	long usedExcessEntries = allocatedExcessEntries - tempData->noAllocatedExcessEntries;

	if (usedBlocks > allocatedBlocks) {
		usedBlocks = allocatedBlocks;
	}
	if (usedExcessEntries > allocatedExcessEntries) {
		usedExcessEntries = allocatedExcessEntries;
	}

	// Display some memory stats, useful for debugging mapping failures.
	float percentFree = 100.0f * (1.0f - static_cast<float>(usedBlocks) / allocatedBlocks);
	float allocatedSizeMiB = scene->localVBA.allocatedSize * sizeof(ITMVoxel) / 1024.0f / 1024.0f;
	printf("[Visible: %6d | Used blocks (primary): %8ld/%ld (%.2f%% free)\n"
			" Used excess list slots: %8ld/%ld | Total allocated size: %.2fMiB]\n",
			tempData->noVisibleBlocks,
			usedBlocks,
			allocatedBlocks,
			percentFree,
			usedExcessEntries,
			allocatedExcessEntries,
			allocatedSizeMiB);

	if (scene->localVBA.lastFreeBlockId < 0) {
		throw std::runtime_error("Invalid free voxel block ID. InfiniTAM has run out of space in "
								 "the Voxel Block Array.");
	}

	if (scene->index.GetLastFreeExcessListId() < 0) {
		throw std::runtime_error("Invalid free excess list slot ID. InfiniTAM has run out of slots "
								 "in the hash table excess list. Consider increasing the size of "
								 "the excess list or the number of buckets.");
	}
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::IntegrateIntoScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;
	if (renderState_vh->noVisibleBlocks == 0) {
		// Our view has no useful data, so there's nothing to allocate. This happens, e.g., when
		// we fuse frames belonging to object instances, in which the actual instance is too far
		// away. Its depth values are over the max depth threshold (and, likely too noisy) and
		// they get ignored, leading to a blank ITMView with nothing new to integrate.
		return;
	}

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	Vector3i *visibleBlockPositions = renderState_vh->GetVisibleBlockPositions();

	dim3 voxelBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 visibleBlockGrid(renderState_vh->noVisibleBlocks);

	WeightParams fusionWeightParams = this->GetFusionWeightParams();

	// These kernels are launched over ALL visible blocks, whose IDs are placed conveniently as the
	// first `renderState_vh->noVisibleEntries` elements of the `visibleEntryIDs` array, which could,
	// in theory, accommodate ALL possible blocks, but usually contains O(10k) blocks.
	if (scene->sceneParams->stopIntegratingAtMaxW) {
		if (trackingState->requiresFullRendering) {
			integrateIntoScene_device<TVoxel, true, false> <<<visibleBlockGrid, voxelBlockSize>>> (
				localVBA, hashTable, visibleBlockPositions, rgb, rgbImgSize, depth, depthImgSize,
				M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW, fusionWeightParams);
		} else {
			integrateIntoScene_device<TVoxel, true, true> <<<visibleBlockGrid, voxelBlockSize>>> (
				localVBA, hashTable, visibleBlockPositions, rgb, rgbImgSize, depth, depthImgSize,
				M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW, fusionWeightParams);
		}
	}
	else {
		if (trackingState->requiresFullRendering) {
			// While developing dynslam, this is the version that is run.
			integrateIntoScene_device<TVoxel, false, false> <<<visibleBlockGrid, voxelBlockSize>>> (
					localVBA, hashTable, visibleBlockPositions, rgb, rgbImgSize, depth, depthImgSize,
						M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW, fusionWeightParams);
		}
		else {
			integrateIntoScene_device<TVoxel, false, true> <<<visibleBlockGrid, voxelBlockSize>>> (
				localVBA, hashTable, visibleBlockPositions, rgb, rgbImgSize, depth, depthImgSize,
						M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW, fusionWeightParams);
		}
	}
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::FullDecay(
		ITMScene<TVoxel, ITMVoxelBlockHash> *scene,
		const ITMRenderState *renderState,
		int minAge,
		int maxWeight
) {
	dim3 voxelBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	// First, we check every bucket and see if it's allocated, populating each index
	// in `visibleBlockGlobalPos` with the block's position, whereby every element in
	// this array corresponds to a VBA element.
	long sdfLocalBlockNum = scene->index.getNumAllocatedVoxelBlocks();
	int noTotalEntries = scene->index.noTotalEntries;
	ITMSafeCall(hipMemset(allocatedBlockPositions_device, 0, sizeof(Vector4s) * sdfLocalBlockNum));

	dim3 hashTableVisitBlockSize(256);
	dim3 hashTableVisitGridSize((noTotalEntries - 1) / hashTableVisitBlockSize.x + 1);

	ITMLib::Engine::findAllocatedBlocks<<<hashTableVisitGridSize, hashTableVisitBlockSize>>>(
			allocatedBlockPositions_device, hashTable, noTotalEntries
	);

	// We now know, for every block allocated in the VBA, whether it's in use, and what its
	// global coordinates are.
	dim3 gridSize(sdfLocalBlockNum);
	ITMSafeCall(hipMemset(locks_device, 0, SDF_BUCKET_NUM * sizeof(int)));
	decayFull_device<TVoxel> <<< gridSize, voxelBlockSize >>> (
			allocatedBlockPositions_device,
			localVBA,
			hashTable,
			minAge,
			maxWeight,
			voxelAllocationList,
			lastFreeBlockId_device,
			locks_device,
			frameIdx,
			((ITMRenderState_VH*)renderState)->GetEntriesVisibleType()
	);

	ITMSafeCall(hipMemcpy(&(scene->localVBA.lastFreeBlockId), lastFreeBlockId_device,
				1 * sizeof(int),
				hipMemcpyDeviceToHost));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::PartialDecay(
		ITMScene<TVoxel, ITMVoxelBlockHash> *scene,
		const ITMRenderState *renderState,
		const VisibleBlockInfo &visibleBlockInfo,
		int minAge,
		int maxWeight
) {
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	ITMSafeCall(hipMemset(locks_device, 0, SDF_BUCKET_NUM * sizeof(int)));

	dim3 voxelBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(static_cast<uint32_t>(visibleBlockInfo.count));
	decay_device<TVoxel> <<< gridSize, voxelBlockSize >>> (
			localVBA,
			hashTable,
			visibleBlockInfo.blockCoords->GetData(MEMORYDEVICE_CUDA),
			minAge,
			maxWeight,
			voxelAllocationList,
			lastFreeBlockId_device,
			locks_device,
			frameIdx,
			((ITMRenderState_VH*)renderState)->GetEntriesVisibleType());

	delete visibleBlockInfo.blockCoords;
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::Decay(
		ITMScene<TVoxel, ITMVoxelBlockHash> *scene,
		const ITMRenderState *renderState,
		int maxWeight,
		int minAge,
		bool forceAllVoxels
) {
	int oldLastFreeBlockId = scene->localVBA.lastFreeBlockId;
	ITMSafeCall(hipMemcpy(lastFreeBlockId_device, &(scene->localVBA.lastFreeBlockId),
						   1 * sizeof(int),
						   hipMemcpyHostToDevice));

	if (forceAllVoxels) {
		FullDecay(scene, renderState, minAge, maxWeight);
	}
	else if (static_cast<long>(frameVisibleBlocks.size()) > minAge) {
		// No full decay, just operate on the voxel blocks seen 'minAge' frames ago.
		VisibleBlockInfo visible = frameVisibleBlocks.front();
		frameVisibleBlocks.pop();

		// Ensure there are voxels to work with. We can often encounter empty frames when
		// reconstructing individual objects which are too far from the camera for any
		// meaningful depth to be estimated, so there's nothing to do for them.
		if (visible.count > 0) {
          PartialDecay(scene, renderState, visible, minAge, maxWeight);
		}
	}

	// This is important for ensuring ITM "knows" about the freed up blocks in the VBA. We also use
	// it for measuring how many voxel blocks were freed.
	ITMSafeCall(hipMemcpy(&(scene->localVBA.lastFreeBlockId), lastFreeBlockId_device,
						   1 * sizeof(int),
						   hipMemcpyDeviceToHost));

	int freedBlockCount = scene->localVBA.lastFreeBlockId - oldLastFreeBlockId;
	totalDecayedBlockCount += freedBlockCount;

	if (freedBlockCount > 0) {
		size_t savings = sizeof(TVoxel) * SDF_BLOCK_SIZE3 * freedBlockCount;
		float savingsMb = (savings / 1024.0f / 1024.0f);

		printf("Found %d candidate blocks to deallocate with weight [%d] or below and age [%d]. "
			   "Saved %.2fMb.\n",
			   freedBlockCount,
			   maxWeight,
			   minAge,
			   savingsMb);
        printf("---------------------------------------------------------------------------\n");
	}
	else {
		printf("Decay process found NO voxel blocks to deallocate.                     \n---------------------------------------------------------------------------\n");
	}
}


template<class TVoxel>
size_t ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::GetDecayedBlockCount() {
	return static_cast<size_t>(totalDecayedBlockCount);
}

// plain voxel array

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMPlainVoxelArray>::ResetScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::IntegrateIntoScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo = scene->index.getIndexData();

	dim3 cudaBlockSize(8, 8, 8);
	dim3 gridSize(
		scene->index.getVolumeSize().x / cudaBlockSize.x,
		scene->index.getVolumeSize().y / cudaBlockSize.y,
		scene->index.getVolumeSize().z / cudaBlockSize.z);

	if (scene->sceneParams->stopIntegratingAtMaxW) {
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, true, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, true, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
	else
	{
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, false, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, false, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::Decay(
		ITMScene<TVoxel, ITMPlainVoxelArray>*, const ITMRenderState*, int, int, bool
) {
  throw std::runtime_error("Map decay is not supported in conjunction with plain voxel arrays, "
						   "only with voxel block hashing.");
}

template<class TVoxel>
size_t ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::GetDecayedBlockCount() {
	throw std::runtime_error("Map decay is not supported in conjunction with plain voxel arrays, "
							 "only with voxel block hashing.");
}


// device functions

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int z = blockIdx.z*blockDim.z+threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * arrayInfo->size.x + z * arrayInfo->size.x * arrayInfo->size.y;
	
	if (stopMaxW) if (voxelArray[locId].w_depth == maxW) return;
//	if (approximateIntegration) if (voxelArray[locId].w_depth != 0) return;

	pt_model.x = (float)(x + arrayInfo->offset.x) * _voxelSize;
	pt_model.y = (float)(y + arrayInfo->offset.y) * _voxelSize;
	pt_model.z = (float)(z + arrayInfo->offset.z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(
			voxelArray[locId],
			pt_model,
			M_d,
			projParams_d,
			M_rgb,
			projParams_rgb,
			mu,
			maxW,
			depth,
			depthImgSize,
			rgb,
			rgbImgSize);
}

// Runs for every block in the visible list => Needs lookup.
template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *localVBA,
										  const ITMHashEntry *hashTable,
										  Vector3i *visibleBlockPositions,
										  const Vector4u *rgb,
										  Vector2i rgbImgSize,
										  const float *depth, Vector2i depthImgSize,
										  Matrix4f M_d, Matrix4f M_rgb,
										  Vector4f projParams_d,
										  Vector4f projParams_rgb,
										  float _voxelSize,
										  float mu,
										  int maxW,
										  ITMLib::Engine::WeightParams weightParams)
{
	Vector3i globalPos;
	bool isFound = false;
	int entryId = findBlock(hashTable, visibleBlockPositions[blockIdx.x], isFound);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;
	int locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	if (!isFound || entryId < 0) {
		if (locId == 0) {
			// Disabled since this is to be expected when resetting volumes of tracks which were being
			// reconstructed but suddenly got lost.
//			printf("FATAL ERROR in integrateIntoScene_device: could not find block in hash map ("
//				   "isFound = %d, entryId = %d, "
//				   "blockIdx.x = %d, locId = %d)! | (%d, %d, %d)\n",
//				   static_cast<int>(isFound),
//				   entryId,
//				   blockIdx.x,
//				   locId,
//				   visibleBlockPositions[blockIdx.x].x,
//				   visibleBlockPositions[blockIdx.x].y,
//				   visibleBlockPositions[blockIdx.x].z
//			);
		}

		return;
	}

	const ITMHashEntry &currentHashEntry = hashTable[entryId];
	globalPos = currentHashEntry.pos.toInt() * SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	Vector4f pt_model;

	if (stopMaxW) if (localVoxelBlock[locId].w_depth == maxW) return;
	if (approximateIntegration) if (localVoxelBlock[locId].w_depth != 0) return;

	pt_model.x = (float)(globalPos.x + x) * _voxelSize;
	pt_model.y = (float)(globalPos.y + y) * _voxelSize;
	pt_model.z = (float)(globalPos.z + z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(localVoxelBlock[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize, weightParams);
}

__global__ void buildHashAllocAndVisibleType_device(
		uchar *entriesAllocType,
		uchar *entriesVisibleType,
		Vector4s *blockCoords,
		const float *depth,
		Matrix4f invM_d,
		Vector4f projParams_d,
		float mu,
		Vector2i _imgSize,
		float _voxelSize,
		ITMHashEntry *hashTable,
		float viewFrustum_min,
		float viewFrustum_max,
		int *locks
) {
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
		projParams_d, mu, _imgSize, _voxelSize, hashTable, viewFrustum_min, viewFrustum_max, locks);
}

// Runs for every block in the visible list => needs lookup.
__global__ void setToType3(uchar *entriesVisibleType,
						   Vector3i *visibleBlockPositions,
						   int noVisibleBlocks,
						   ITMHashEntry *hashTable)
{
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId >= noVisibleBlocks) {
		return;
	}

	bool isFound = false;
	int hashIdx = findBlock(hashTable, visibleBlockPositions[entryId], isFound);

	if (! isFound || hashIdx < 0) {
		// The block (which was in sight last frame) was cleared out by the decay. Not an error, but
		// may lead to artifacts in the map. This can happen if the decay "catches up" with the
		// active reconstruction.
//      int hashVal = hashIndex(visibleBlocks[entryId]);
//		if (hashVal % 100 < 40) {
//			printf("WARNING in setToType3 visibleBlocks[%d]: (isFound = %d, hashIdx = %d"
//						   ")! | (%d, %d, %d) @ hashVal = %d\n",
//				   entryId,
//				   static_cast<int>(isFound),
//				   hashIdx,
//				   visibleBlocks[entryId].x,
//				   visibleBlocks[entryId].y,
//				   visibleBlocks[entryId].z,
//				   hashVal
//			);
//		}
	}
	else {
		entriesVisibleType[hashIdx] = 3;
	}
}

__global__ void allocateVoxelBlocksList_device(
		int *voxelAllocationList, int *excessAllocationList,
		ITMHashEntry *hashTable, int noTotalEntries,
		AllocationTempData *allocData,
		uchar *entriesAllocType, uchar *entriesVisibleType,
		Vector4s *blockCoords,
		int currentFrame
) {
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;

	switch (entriesAllocType[targetIdx])
	{
		case 0: // TODO(andrei): Could we please use constants/enums/defines for these values?
			// 0 == Invisible block.
		break;

	case 1:
		// 1 == block visible and needs allocation, fits in the ordered list.
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			ITMHashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;
			hashEntry.allocatedTime = currentFrame;

			hashTable[targetIdx] = hashEntry;
		}
		else
		{
			// TODO(andrei): Handle this better.
			printf("WARNING: No more room in VBA! vbaIdx became %d.\n", vbaIdx);
			printf("exlIdx is %d.\n", allocData->noAllocatedExcessEntries);
		}
		break;

	case 2:
		// 2 == block visible and needs allocation in the excess list
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		exlIdx = atomicSub(&allocData->noAllocatedExcessEntries, 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			ITMHashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;
			hashEntry.allocatedTime = currentFrame;

			int exlOffset = excessAllocationList[exlIdx];

			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			hashTable[SDF_BUCKET_NUM + exlOffset] = hashEntry; //add child to the excess list

			entriesVisibleType[SDF_BUCKET_NUM + exlOffset] = 1; //make child visible
		}
		else
		{
			// TODO(andrei): Handle this better. We could probably get away with just looking at
			// noAllocatedVoxelEntries and noAllocatedExcessEntries after the kernel completes.
			if (vbaIdx >= 0)
			{
				printf("WARNING: Could not allocate in excess list! There was still room in the main VBA, "
						   "but exlIdx = %d! Consider increasing the overall hash table size, or at least the "
						   "bucket size.\n", exlIdx);
			}
			else if(exlIdx)
			{
				printf("WARNING: Tried to allocate in excess list, but failed because the main VBA is "
							 "full. vbaIdx = %d\n", vbaIdx);
			}
			else
			{
				printf("WARNING: No more room in VBA or in the excess list! vbaIdx became %d.\n", vbaIdx);
				printf("exlIdx is %d.\n", exlIdx);
			}
		}
		break;

		default:
			printf("Unexpected alloc type: %d\n", static_cast<int>(entriesAllocType[targetIdx]));
		break;
	}
}

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, /*int *noAllocatedVoxelEntries,*/ uchar *entriesVisibleType)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx;
	int hashEntry_ptr = hashTable[targetIdx].ptr;

	if (entriesVisibleType[targetIdx] > 0 && hashEntry_ptr == -1) //it is visible and has been previously allocated inside the hash, but deallocated from VBA
	{
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		if (vbaIdx >= 0) hashTable[targetIdx].ptr = voxelAllocationList[vbaIdx];
	}
}

// Gets called for every entry in the hash table (ordered + excess) => no lookup.
template<bool useSwapping>
__global__ void buildVisibleList_device(
		ITMHashEntry *hashTable,
		ITMHashSwapState *swapStates,
		int noTotalEntries,
        Vector3i *visibleBlockPositions,
		AllocationTempData *allocData,
		uchar *entriesVisibleType,
		Matrix4f M_d,
		Vector4f projParams_d,
		Vector2i depthImgSize,
		float voxelSize
) {
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned char hashVisibleType = 0;

	__shared__ bool shouldPrefix;
	shouldPrefix = false;
	__syncthreads();

	if (targetIdx < noTotalEntries) {
		hashVisibleType = entriesVisibleType[targetIdx];
		const ITMHashEntry & hashEntry = hashTable[targetIdx];

		// i.e., previously seen
		if (hashVisibleType == 3)
		{
			bool isVisibleEnlarged, isVisible;

			if (useSwapping)
			{
				checkBlockVisibility<true>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, 	depthImgSize);
				if (!isVisibleEnlarged) hashVisibleType = 0;
			} else {
				checkBlockVisibility<false>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, 	depthImgSize);
				if (!isVisible) hashVisibleType = 0;
			}
			entriesVisibleType[targetIdx] = hashVisibleType;
		}

		// If we've seen the block last frame, and it's still visible, keep it.
		if (hashVisibleType > 0) shouldPrefix = true;

		if (useSwapping) {
			if (hashVisibleType > 0 && swapStates[targetIdx].state != 2) swapStates[targetIdx].state = 1;
		}
    }

	__syncthreads();

	// Computes the correct offsets for the visible blocks in the output visibleBlockPositions array.
	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType > 0,
												  &allocData->noVisibleBlocks,
												  blockDim.x * blockDim.y,
												  threadIdx.x);
		if (offset != -1) {
			// -1 is returned for entries which contribute a 'false' so don't need to be written to
			// the visible list.
			visibleBlockPositions[offset] = hashTable[targetIdx].pos.toInt();
		}
	}

#if 0
	// "active list": blocks that have new information from depth image
	// currently not used...
	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType == 1, noActiveEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) activeEntryIDs[offset] = targetIdx;
	}
#endif
}

/// \brief Deletes a block from the hash table and de-allocates its VBA entry.
/// \param hashTable
/// \param blockPos             The position of the block in the voxel grid, i.e., the key.
/// \param locks                Array used for locking in order to prevent data races when
///                             attempting to delete multiple elements with the same key.
/// \param voxelAllocationList  List of free voxels.
/// \param lastFreeBlockId      Index in the voxel allocation list (free list).
/// \param entriesVisibleType   For every entry in the hash table, keeps track of whether it's
///                             visible in the last/current frame.
///
/// \note Does not support swapping.
template<class TVoxel, bool paranoid=false>
__device__
void deleteBlock(
		ITMHashEntry *hashTable,
		Vector3i blockPos,
		int *locks,
		int *voxelAllocationList,
		int *lastFreeBlockId,
		uchar *entriesVisibleType
) {
	int keyHash = hashIndex(blockPos);

	// Lock the bucket for the operation, to ensure the lists stay consistent
	int status = atomicExch(&locks[keyHash], BUCKET_LOCKED);
	if (status != BUCKET_UNLOCKED) {
		printf("Contention on bucket of hash value %d. Not going further with deletion of block "
					   "(%d, %d, %d).\n", keyHash, blockPos.x, blockPos.y, blockPos.z);
		return;
	}

	bool isFound = false;
	int outBlockIdx = -1;
	int outPrevBlockIdx = -1;
	findVoxel(hashTable, blockPos, 0, isFound, outBlockIdx, outPrevBlockIdx);

	bool isExcess = (outBlockIdx >= SDF_BUCKET_NUM);

	// Paranoid sanity checks
	if (paranoid) {
		if (outPrevBlockIdx == -1) {
			if (isExcess) {
				printf("\n[ERROR] Found entity in excess list with no previous element (%d, %d, %d)!\n",
					   blockPos.x,
					   blockPos.y,
					   blockPos.z);
			}
		} else {
			if (!isExcess) {
				printf("\n[ERROR] Found entity in bucket list with a previous guy!\n");
			}
		}

		if (!isFound || outBlockIdx < 0) {
			if (blockPos.x % 10 == 3) {
				printf("\n\nFATAL ERROR: sanity check failed in 'decay_device' voxel (block) "
							   "found = %d, outBlockIdx = %d (%d, %d, %d) ; %s.\n",
					   static_cast<int>(isFound),
					   outBlockIdx,
					   blockPos.x,
					   blockPos.y,
					   blockPos.z,
					   isExcess ? "excess" : "non-excess"
				);
			}
			atomicExch(&locks[keyHash], BUCKET_UNLOCKED);
			return;
		}
	}

	// First, deallocate the VBA slot.
	int freeListIdx = atomicAdd(&lastFreeBlockId[0], 1);
	voxelAllocationList[freeListIdx + 1] = hashTable[outBlockIdx].ptr;

	// TODO(andrei): Update excess freelist! (Should work without doing it but leak excess slots.)
	// If updating the excess free list, make sure you also sync back the proper 'last excess slot
	// index' index.

	// Second, clear out the hash table entry, and do bookkeeping for buckets with more than one element.
	if (outPrevBlockIdx == -1) {
		// In the ordered list
		if (hashTable[outBlockIdx].offset >= 1) {
			// In the ordered list, with a successor.
			long nextIdx = SDF_BUCKET_NUM + hashTable[outBlockIdx].offset - 1;
			hashTable[outBlockIdx] = hashTable[nextIdx];

			entriesVisibleType[outBlockIdx] = entriesVisibleType[nextIdx];
			entriesVisibleType[nextIdx] = 0;

			// Free up the slot we just copied into the main VBA, in case there's still pointers
			// to it in the visible list from some to-be-decayed frame.
			// [RIP] Not doing this can mean the zombie block gets detected as valid in the future,
			// even though it's in the excess area but nobody is pointing at it.
			hashTable[nextIdx].offset = 0;
			hashTable[nextIdx].ptr = -2;
		}
		else {
			// In the ordered list, and no successor.
			hashTable[outBlockIdx].ptr = -2;
			entriesVisibleType[outBlockIdx] = 0;
		}
	}
	else {
		// In the excess list with a successor or not.
		hashTable[outPrevBlockIdx].offset = hashTable[outBlockIdx].offset;
		hashTable[outBlockIdx].offset = 0;
		hashTable[outBlockIdx].ptr = -2;

		entriesVisibleType[outPrevBlockIdx] = entriesVisibleType[outBlockIdx];
		entriesVisibleType[outBlockIdx] = 0;
	}

	// Release the lock.
	atomicExch(&locks[keyHash], BUCKET_UNLOCKED);
}


/// \brief Looks up a voxel, determines if it should be decayed, and then automatically deletes the
///        blocks that become empty in the process.
template<class TVoxel>
__device__
void decayVoxel(
		Vector3i blockPos,
		int locId,
		TVoxel *localVBA,				// could wrap in HashMap struct
		ITMHashEntry *hashTable,		// could wrap
		int minAge,
		int maxWeight,
		int *voxelAllocationList,		// could wrap
		int *lastFreeBlockId,			// could wrap
		int *locks,
		int currentFrame,
		uchar *entriesVisibleType		// could wrap
) {
	bool isFound = false;
	int blockHashIdx = -1;
	int blockPrevHashIdx = -1;

	int voxelIdx = findVoxel(hashTable, blockPos, locId, isFound, blockHashIdx, blockPrevHashIdx);

	if (-1 == blockHashIdx) {
		if (locId == 0) {
			printf("ERROR: could not find bucket for (%d, %d, %d) @ hash ID %d.\n",
				   blockPos.x, blockPos.y, blockPos.z, hashIndex(blockPos));
		}
		return;
	}

	bool emptyVoxel = false;
	bool safeToClear = true;
	int age = currentFrame - hashTable[blockHashIdx].allocatedTime;
	if (age < minAge) {
		// Important corner case: when we had a block in the visible list, but it got deleted in
		// a previous decay pass, and ended up also getting reallocated (and thus the old ID in
		// the visible list was pointing to the wrong thing).
		safeToClear = false;
	}

	if (safeToClear) {
		// The SDF limit it EXPERIMENTAL and enabling it may be to aggressive when applied on a
		// per-voxel basis.
//		float sdfLim = 10.00f;
//		bool isNoisy = (localVBA[voxelIdx].w_depth <= maxWeight || localVBA[voxelIdx].sdf > sdfLim);
		bool isNoisy = (localVBA[voxelIdx].w_depth <= maxWeight);
		if (isNoisy && localVBA[voxelIdx].w_depth > 0) {
			localVBA[voxelIdx].reset();
			emptyVoxel = true;
		}

		if (localVBA[voxelIdx].w_depth == 0) {
			emptyVoxel = true;
		}
	}

	// Count the empty voxels in the block, to determine if it's empty
	// TODO(andrei): Try summing all the weights and empty == weightSum < k (==3-10).
	static const int voxelsPerBlock = SDF_BLOCK_SIZE3;
	__shared__ int countBuffer[voxelsPerBlock];
	countBuffer[locId] = static_cast<int>(emptyVoxel);
	__syncthreads();

	// Block-level sum for counting non-empty voxels in this block.
	blockReduce(countBuffer, voxelsPerBlock, locId);
	__syncthreads();

	int emptyVoxels = countBuffer[0];
	bool emptyBlock = (emptyVoxels == voxelsPerBlock);

	if (locId == 0 && emptyBlock && safeToClear) {
		deleteBlock<TVoxel>(hashTable,
							blockPos,
							locks,
							voxelAllocationList,
							lastFreeBlockId,
							entriesVisibleType);
	}
}

/// This kernel runs per-voxel, just like 'decayFull_device', for every block in the visible list
/// (so we need to perform lookups).
template<class TVoxel>
__global__
void decay_device(TVoxel *localVBA,
				  ITMHashEntry *hashTable,
				  Vector3i *visibleBlockPositions,
				  int minAge,
				  int maxWeight,
				  int *voxelAllocationList,
				  int *lastFreeBlockId,
				  int *locks,
				  int currentFrame,
				  uchar *entriesVisibleType
) {
	// Note: there are no range checks because we launch exactly as many threads as we need.
	// The local offset of the voxel in the current block.
	int locId = threadIdx.x + threadIdx.y * SDF_BLOCK_SIZE + threadIdx.z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	bool isFound = false;
	int hashIdx = findBlock(hashTable, visibleBlockPositions[blockIdx.x], isFound);

	if (!isFound || hashIdx < 0) {
		// The block was already de-allocated.
		return;
	}

	const ITMHashEntry &currentHashEntry = hashTable[hashIdx];

	Vector3i blockGridPos = currentHashEntry.pos.toInt();
	decayVoxel<TVoxel>(blockGridPos, locId, localVBA, hashTable, minAge, maxWeight,
					   voxelAllocationList, lastFreeBlockId, locks, currentFrame, entriesVisibleType);
}


template<class TVoxel>
__global__
void decayFull_device(
		const Vector4s *usedBlockPositions,
		TVoxel *localVBA,
		ITMHashEntry *hashTable,
		int minAge,
		int maxWeight,
		int *voxelAllocationList,
		int *lastFreeBlockId,
		int *locks,
		int currentFrame,
		uchar *entriesVisibleType
) {
	const int voxelBlockIdx = blockIdx.x;
	const Vector4s blockGridPos_4s = usedBlockPositions[voxelBlockIdx];

	if (blockGridPos_4s.w == 0) {
		// A zero means no hash table entry points to this block.
		return;
	}

	// Note: this also acts as the key for the voxel block hash.
	const Vector3i blockPos = Vector3i(blockGridPos_4s.x, blockGridPos_4s.y, blockGridPos_4s.z);
	int locId = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;

	decayVoxel<TVoxel>(blockPos, locId, localVBA, hashTable, minAge, maxWeight, voxelAllocationList,
					 lastFreeBlockId, locks, currentFrame, entriesVisibleType);
}


template class ITMLib::Engine::ITMSceneReconstructionEngine_CUDA<ITMVoxel, ITMVoxelIndex>;

