#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include <iostream>
#include "ITMMeshingEngine_CUDA.h"
#include "../../DeviceAgnostic/ITMMeshingEngine.h"
#include "ITMCUDAUtils.h"

#include "../../../../ORUtils/CUDADefines.h"

template<class TVoxel>
__global__ void meshScene_device(ITMMesh::Triangle *triangles, unsigned int *noTriangles_device, float factor, int noTotalEntries,
	int noMaxTriangles, const Vector4s *visibleBlockGlobalPos, const TVoxel *localVBA, const ITMHashEntry *hashTable);

using namespace ITMLib::Engine;

template<class TVoxel>
ITMMeshingEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMMeshingEngine_CUDA(long sdfLocalBlockNum)
	: sdfLocalBlockNum(sdfLocalBlockNum)
{
	ITMSafeCall(hipMalloc((void**)&visibleBlockGlobalPos_device, sdfLocalBlockNum * sizeof(Vector4s)));
	ITMSafeCall(hipMalloc((void**)&noTriangles_device, sizeof(unsigned int)));
}

template<class TVoxel>
ITMMeshingEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMMeshingEngine_CUDA(void)
{
	ITMSafeCall(hipFree(visibleBlockGlobalPos_device));
	ITMSafeCall(hipFree(noTriangles_device));
}

/// \brief Hacky operator for easily displaying CUDA dim3 objects.
std::ostream& operator<<(std::ostream &out, dim3 dim) {
	out << "[" << dim.x << ", " << dim.y << ", " << dim.z << "]";
	return out;
}

template<class TVoxel>
void ITMMeshingEngine_CUDA<TVoxel, ITMVoxelBlockHash>::MeshScene(ITMMesh *mesh, const ITMScene<TVoxel, ITMVoxelBlockHash> *scene)
{
	// TODO-LOW(andrei): This doesn't work if swapping is enabled. That is, it only saves the active
	// mesh, and doesn't attempt to somehow stream all the blocks which have been swapped out to
	// RAM. (It would *not* be trivial to extend the meshing engine to support this IMHO.)

	ITMMesh::Triangle *triangles = mesh->triangles->GetData(MEMORYDEVICE_CUDA);
	const TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	const ITMHashEntry *hashTable = scene->index.GetEntries();

	int noMaxTriangles = mesh->noMaxTriangles, noTotalEntries = scene->index.noTotalEntries;
	float factor = scene->sceneParams->voxelSize;

	ITMSafeCall(hipMemset(noTriangles_device, 0, sizeof(unsigned int)));
	ITMSafeCall(hipMemset(visibleBlockGlobalPos_device, 0, sizeof(Vector4s) * sdfLocalBlockNum));

	{ // identify used voxel blocks
		dim3 cudaBlockSize(256); 
		dim3 gridSize((int)ceil((float)noTotalEntries / (float)cudaBlockSize.x));
		findAllocatedBlocks << <gridSize, cudaBlockSize >> >(visibleBlockGlobalPos_device, hashTable, noTotalEntries);
	}

	{ // mesh used voxel blocks
		dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
		dim3 gridSize(sdfLocalBlockNum / 16, 16);

		meshScene_device<TVoxel> << <gridSize, cudaBlockSize >> >(
				triangles,
				noTriangles_device,
				factor,
				noTotalEntries,
				noMaxTriangles,
				visibleBlockGlobalPos_device,
				localVBA,
				hashTable);

		ITMSafeCall(hipMemcpy(
				&mesh->noTotalTriangles,
				noTriangles_device,
				1 * sizeof(unsigned int),
				hipMemcpyDeviceToHost));
		printf("Meshing done: %d/%d triangles in mesh.\n", mesh->noTotalTriangles, mesh->noMaxTriangles);
	}
}

template<class TVoxel>
ITMMeshingEngine_CUDA<TVoxel,ITMPlainVoxelArray>::ITMMeshingEngine_CUDA(long sdfLocalBlockNum)
		: sdfLocalBlockNum(sdfLocalBlockNum)
{}

template<class TVoxel>
ITMMeshingEngine_CUDA<TVoxel,ITMPlainVoxelArray>::~ITMMeshingEngine_CUDA(void) 
{}

template<class TVoxel>
void ITMMeshingEngine_CUDA<TVoxel, ITMPlainVoxelArray>::MeshScene(ITMMesh *mesh, const ITMScene<TVoxel, ITMPlainVoxelArray> *scene)
{}

__global__ void ITMLib::Engine::findAllocatedBlocks(
		Vector4s *visibleBlockGlobalPos,
		const ITMHashEntry *hashTable,
		int noTotalEntries
) {
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId > noTotalEntries - 1) return;

	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	// If this bucket is not unused (ptr < -1), and not swapped out (ptr == -1), we are interested
	// in it in the next stage.
	if (currentHashEntry.ptr >= 0) {
		// visibleBlockGlobalPos maps each VBA entry to the block's position in 3D. If a VBA is not
		// referenced, its 'w' stays 0. Neat!
		visibleBlockGlobalPos[currentHashEntry.ptr] = Vector4s(
				currentHashEntry.pos.x, currentHashEntry.pos.y, currentHashEntry.pos.z, 1);
	}
}

template<class TVoxel>
__global__ void meshScene_device(ITMMesh::Triangle *triangles, unsigned int *noTriangles_device, float factor, int noTotalEntries, 
	int noMaxTriangles, const Vector4s *visibleBlockGlobalPos, const TVoxel *localVBA, const ITMHashEntry *hashTable)
{
	const Vector4s globalPos_4s = visibleBlockGlobalPos[blockIdx.x + gridDim.x * blockIdx.y];

	if (globalPos_4s.w == 0) return;

	Vector3i globalPos = Vector3i(globalPos_4s.x, globalPos_4s.y, globalPos_4s.z) * SDF_BLOCK_SIZE;

	Vector3f vertList[12];

	Vector3i localPos = Vector3i(threadIdx.x, threadIdx.y, threadIdx.z);
	int cubeIndex = buildVertList(vertList, globalPos, localPos, localVBA, hashTable);

	if (cubeIndex < 0) return;

	for (int i = 0; triangleTable[cubeIndex][i] != -1; i += 3)
	{
		int triangleId = atomicAdd(noTriangles_device, 1);

		if (triangleId < noMaxTriangles - 1)
		{
			Vector3f p0 = vertList[triangleTable[cubeIndex][i]];
			Vector3f p1 = vertList[triangleTable[cubeIndex][i + 1]];
			Vector3f p2 = vertList[triangleTable[cubeIndex][i + 2]];
			triangles[triangleId].p0 = p0 * factor;
			triangles[triangleId].p1 = p1 * factor;
			triangles[triangleId].p2 = p2 * factor;

			Vector3f c0 =
					VoxelColorReader<TVoxel::hasColorInformation, TVoxel, ITMVoxelBlockHash>::interpolate3(
							localVBA,
							hashTable,
                            p0);
			Vector3f c1 =
					VoxelColorReader<TVoxel::hasColorInformation, TVoxel, ITMVoxelBlockHash>::interpolate3(
							localVBA,
							hashTable,
                            p1);
			Vector3f c2 =
					VoxelColorReader<TVoxel::hasColorInformation, TVoxel, ITMVoxelBlockHash>::interpolate3(
							localVBA,
							hashTable,
                            p2);

			triangles[triangleId].c0 = c0;
			triangles[triangleId].c1 = c1;
			triangles[triangleId].c2 = c2;
		}
	}
}

template class ITMLib::Engine::ITMMeshingEngine_CUDA<ITMVoxel, ITMVoxelIndex>;
